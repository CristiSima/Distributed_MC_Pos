#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>


#define MODE 1


#include "base.h"

#if !defined(NUM_BLOCKS)
    #define NUM_BLOCKS 6
#endif

#if !defined(NUM_THREADS)
    #define NUM_THREADS 768
#endif

#if !defined(THREAD_OVERLOAD)
    #define THREAD_OVERLOAD 1
#endif

#if !defined(WORKER_OFFSET)
    #define WORKER_OFFSET 0
#endif

#if !defined(TOTAL_WORKERS)
    #define TOTAL_WORKERS (NUM_THREADS*NUM_BLOCKS)
#endif

__global__ void print_from_gpu(void) {
    int sum=0;
    int y=10;
    int dx=threadIdx.x*WORLD_LIMIT;
    for(int x=0;x<WORLD_LIMIT;x++) for(int z=0;z<WORLD_LIMIT;z++)
    {
        if(check_pos(x+dx, y, z))
        {
            printf("FOUND @ %d %d %d\n", x+dx, y, z);
            sum++;
        }
    }
    if(sum!=0)
	printf("Hello World! from thread [%d,%d]  %d\n", threadIdx.x, blockIdx.x, sum);
}


__global__ void cool_search_wraper()
{
    for(int id_overload=0;id_overload<THREAD_OVERLOAD;id_overload++)
        cool_search(WORKER_OFFSET, TOTAL_WORKERS, threadIdx.x+blockIdx.x*blockDim.x+ id_overload*(blockDim.x*gridDim.x));
}



int main(void) {
	printf("GPU[%3d, %3d] with limit %8d\n", NUM_BLOCKS, NUM_THREADS, WORLD_LIMIT);
    fflush(stdout);

    #define cool_search_standalone(workers) cool_search_wraper<<<1,workers>>>(0, workers);
    #define cool_search_test(offset, total_workers) cool_search_wraper<<<1,1>>>(offset, total_workers);
    cool_search_wraper<<<NUM_BLOCKS, NUM_THREADS>>>();

    hipError_t cuda_error=hipGetLastError();
    if(cuda_error!=hipSuccess)
        printf("%s\n%s\n", hipGetErrorName(cuda_error), hipGetErrorString(cuda_error));
    // cool_search_standalone(768);
    // cool_search_standalone(768);
    // cool_search_standalone(1);
    // cool_search_test(1, 2);
    // cool_search_wraper<<<1,768>>>(0, 768);
	// print_from_gpu<<<1,768>>>();
	// print_from_gpu<<<1,768>>>();
	// print_from_gpu<<<1,400>>>();
	hipDeviceSynchronize();
    printf("GPU DONE\n");
    return 0;
}

// nvcc main.cu      -o main; $runTime = (Measure-Command { .\main.exe | Out-Default }).TotalSeconds; Write-Host "GPU RunTime:  " $runTime s
// g++  main.cpp -O2 -o main; $runTime = (Measure-Command { .\main.exe | Out-Default }).TotalSeconds; Write-Host "CPU RunTime:  " $runTime s
